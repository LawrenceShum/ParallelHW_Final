# include "../include/SHTimer.cuh"

SHTimer::SHTimer() : timeElapsed(0.0f)
{
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
}
SHTimer::~SHTimer()
{
	hipEventDestroy(start);
	hipEventDestroy(stop);
}
void SHTimer::startTimer()
{
	hipEventRecord(start, 0);
}
float SHTimer::stopTimer()
{
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	checkCudaErrors(hipEventElapsedTime(&timeElapsed, start, stop));
	return timeElapsed;
}